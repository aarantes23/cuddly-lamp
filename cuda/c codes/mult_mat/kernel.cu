#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <>
#include <iostream>
#include "getRealTime.h"
#define tamBloco 16

__global__ void matrixMult(float *A, float *B, float *C, int N)
{
	//Multiplica��o de matrix C=A*B de tamanho NxN
	//Cada thread computa um �nico elemento da matrix C
	int linha = blockIdx.y*blockDim.y + threadIdx.y;
	int coluna = blockIdx.x*blockDim.x + threadIdx.x;
	float sum = 0.f;
	for (int n = 0; n < N; ++n)
		sum += A[linha*N + n] * B[n*N + coluna];
	C[linha*N + coluna] = sum;
	__syncthreads();
}
int main()
{
	double inicio, fim;
	int N, K;
	K = 1000;
	N = K * tamBloco;
	//Criando matrizes no host
	float *hA, *hB, *hC;
	hA = new float[N*N];
	hB = new float[N*N];
	hC = new float[N*N];
	//Iniciando matrizes
	for (int j = 0; j < N; j++) {
		for (int i = 0; i < N; i++) {
			hA[j*N + i] = 2.f*(j + i);
			hB[j*N + i] = 1.f*(j - i);
		}
	}
	//Criando as matrizes no device
	float *dA, *dB, *dC;
	hipMalloc((void**)&dA, N*N * sizeof(float));
	hipMalloc((void**)&dB, N*N * sizeof(float));
	hipMalloc((void**)&dC, N*N * sizeof(float));
	dim3 threadBlock(tamBloco, tamBloco);//threads = 16x16
	dim3 grid(K, K);//blocos = KxK

	//Copiando as matrizes do host para o device
	hipMemcpy(dA, hA, N, hipMemcpyHostToDevice);
	hipMemcpy(dB, hB, N, hipMemcpyHostToDevice);

	inicio = getRealTime();	
	//execu��o do kernel
	matrixMult << <grid, threadBlock >> > (dA, dB, dC, N);
	fim = getRealTime();
	//Copia o resultado do device para cpu
	hipMemcpy(hC, dC, N*N * sizeof(float), hipMemcpyDeviceToHost);
	hipDeviceSynchronize();
	hipDeviceReset();

	
	fprintf(stderr, "Tamanho = %d\nTempo gasto = %lf segundos\n", K,(fim - inicio));
	return 0;
}