#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <>
#include <stdio.h>
#include "getRealTime.h"
// #include "fibonnaci_algorithm.h"

const int qtdNum = 500;
__device__ unsigned long long recursiveFibonnaci(int nSize) {
	switch (nSize) {
	case 0:
		return 0;
		break;
	case 1:
		return 1;
		break;
	default:
		return recursiveFibonnaci((nSize - 2)) + recursiveFibonnaci((nSize - 1));
	}
	
}
__global__ void kernel(int *d_sequenciaFib) {
	int i = threadIdx.x;
	d_sequenciaFib[i] = recursiveFibonnaci(i);
	__syncthreads();
}
int main()
{
	double inicio, fim, delta;
	int recursiveFibonnaci[qtdNum];
	int *d_sequenciaFib;

	inicio = getRealTime();

	hipMalloc((void**)&d_sequenciaFib, qtdNum * sizeof(int));
	kernel << <1, qtdNum >> > (d_sequenciaFib);
	hipMemcpy(recursiveFibonnaci, d_sequenciaFib, qtdNum * sizeof(int),
		hipMemcpyDeviceToHost);
	hipDeviceSynchronize();
	inicio = getRealTime();
	for (int i = 0; i < qtdNum; ++i) {
		unsigned long long fibonnaci_ = recursiveFibonnaci[i];
	}

	fim = getRealTime();

	hipDeviceReset();
	fprintf(stderr, "Tempo gasto = %lf segundos\n", (fim - inicio));
	return 0;
}